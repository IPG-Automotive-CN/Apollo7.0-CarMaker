#include <stdio.h>
#include <pthread.h>
#include <hip/hip_runtime.h>

#define N_THREAD 2

void *thread_run(void *pp)
{
    int *p=(int*)pp;
    
    int tid=p[0];
    int id;
		
char data[]={'c'}; 
    hipGetDevice(&id);
    printf("%d idev=%d\n",tid,id);
    unsigned char* m = ( unsigned char*)malloc(sizeof(char)*3);
    
    hipMemcpy(m, data, 1, hipMemcpyDefault);
    
    pthread_exit(NULL);
}

int main(int argc,char *argv[])
{
    pthread_t tid[2];
    int i;
    int ii[2];
    
    int a = hipSetDevice(0);
    if(a == hipSuccess){
    	printf("cuda success 1\n");
	}    
    for (i=0;i<N_THREAD;++i)
    {
        ii[i]=i;
        pthread_create(&tid[i],NULL,thread_run,&ii[i]);
    }
    
    for (i=0;i<N_THREAD;++i)
    {
        pthread_join(tid[i],NULL);
    }
    int b = hipSetDevice(0);
    if(b == hipSuccess){
    	printf("cuda success 2\n");
	}    
}
